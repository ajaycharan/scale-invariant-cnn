#include "hip/hip_runtime.h"
extern "C" {
	#include "lua.h"
	#include "lualib.h"
	#include "lauxlib.h"
}

#include "luaT.h"
//#include "THC.h"
#include "THCUNN.h"
#include <stdio.h>
#include <assert.h>

THCState* getCutorchState(lua_State* L) {
	lua_getglobal(L, "cutorch");
	lua_getfield(L, -1, "getState");
	lua_call(L, 0, 1);
	THCState *state = (THCState*) lua_touserdata(L, -1);
	lua_pop(L, 2);
	return state;
}

static const int NTHREADS = 32;

__global__ void cunn_ClassNLLCriterionWithIgnore_updateOutput_kernel1(float *output,
    float *total_weight,
    float *input,
    float *target,
    float *weights,
    int size_average,
    int n_classes) {
  assert(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0);

  // TODO: T4951791 Reuse code between updateOutput_kernel1 and
  // updateOutput_kernel.

  int t = (int)*target - 1;
  assert(t >= 0 && t < n_classes);
  float cur_weight = weights ? weights[t] : 1.0f;
  *output = -cur_weight * input[t];
  *total_weight = cur_weight;
  if (size_average && *total_weight > 0) {
    *output /= *total_weight;
  }
}

__global__ void cunn_ClassNLLCriterionWithIgnore_updateOutput_kernel(float *output,
    float *total_weight, 
    float *input,
    float *target,
    float *weights,
    int size_average,
    int nframe,
    int ndim,
    int n_classes) {
  __shared__ float shInputs[NTHREADS], acc_weight[NTHREADS];
  int i, t;
  float cur_weight;

  shInputs[threadIdx.x] = 0.0f;
  acc_weight[threadIdx.x] = 0.0f;
  for (i = threadIdx.x; i < nframe; i += NTHREADS) {
    // check for ignore label
    t = (int)target[i] - 1;
    if (t < 0)
      continue;
    assert(t >= 0 && t < n_classes);
    cur_weight = weights ? weights[t] : 1.0f;
    shInputs[threadIdx.x] -= input[i * ndim + t] * cur_weight;
    acc_weight[threadIdx.x] += cur_weight;
  }
  __syncthreads();

  // TODO: T4951791 Reuse code between updateOutput_kernel1 and
  // updateOutput_kernel

  if (threadIdx.x == 0) {
    *output = *total_weight = 0;
    for (i = 0; i < NTHREADS; ++i){
      *output += shInputs[i];
      *total_weight += acc_weight[i];
    }
    if (size_average && *total_weight > 0) {
      *output /= *total_weight;
    }
  }
}

__global__ void cunn_ClassNLLCriterionWithIgnore_updateGradInput_kernel1(
    float* gradInput,
    float* weights,
    float* target,
    float* total_weight,
    int size_average,
    int n_classes)
{
  if (*total_weight <= 0) {
    return;
  }
  float norm = size_average ? (1.0f / *total_weight) : 1.0f;
  int t = (int)*target - 1;
  assert(t >= 0 && t < n_classes);
  gradInput[t] = -(weights ? weights[t] : 1.0f) * norm;
}

__global__ void cunn_ClassNLLCriterionWithIgnore_updateGradInput_kernel(
    float *gradInput,
    float *target,
    float *weights,
    float *total_weight,
    int size_average,
    int nframe,
    int ndim,
    int n_classes)
{
  if (*total_weight <= 0) {
    return;
  }
  int i, t;
  float norm = size_average ? (1.0f / *total_weight) : 1.0f;

  for (i = threadIdx.x; i < nframe; i += NTHREADS) {
    t = (int)target[i] - 1;
    if (t < 0)
      continue;
    assert(t >= 0 && t < n_classes);
    gradInput[i * ndim + t] = -(weights ? weights[t] : 1.0f) * norm;
  }
}

int CudaClassNLLCriterionWithIgnore_updateOutput(THCState *state, THCudaTensor *input, THCudaTensor *target, THCudaTensor *output, bool sizeAverage, THCudaTensor *weights, THCudaTensor *total_weight) {
  if (THCudaTensor_nDimension(state, target) > 1) {
    THError("multi-target not supported");
  }

  int n_dims = THCudaTensor_nDimension(state, input);
  int n_classes = THCudaTensor_size(state, input, n_dims - 1);

  if (weights) {
    THAssert(THCudaTensor_checkGPU(
          state, 5, input, target, weights, output, total_weight
          ));
  } else {
    THAssert(THCudaTensor_checkGPU(
          state, 4, input, target, output, total_weight
          ));
  }

  if (THCudaTensor_nDimension(state, input) > 2) {
    THArgCheck(0, 2, "vector or matrix expected");
  }

  input = THCudaTensor_newContiguous(state, input);
  weights = weights ? THCudaTensor_newContiguous(state, weights) : NULL;
  target = THCudaTensor_newContiguous(state, target);

  float *input_data = THCudaTensor_data(state, input);
  float *weights_data = weights ? THCudaTensor_data(state, weights) : NULL;
  float *target_data = THCudaTensor_data(state, target);
  float *output_data = THCudaTensor_data(state, output);
  float *total_weight_data = THCudaTensor_data(state, total_weight);

  if (THCudaTensor_nDimension(state, input) == 1) {
    cunn_ClassNLLCriterionWithIgnore_updateOutput_kernel1 
      <<<1, 1, 0, THCState_getCurrentStream(state)>>>(
          output_data,
          total_weight_data,
          input_data,
          target_data,
          weights_data,
          sizeAverage,
          n_classes
          );

  } else if (THCudaTensor_nDimension(state, input) == 2) {
    cunn_ClassNLLCriterionWithIgnore_updateOutput_kernel
      <<<1, NTHREADS, 0, THCState_getCurrentStream(state)>>>(
          output_data,
          total_weight_data, 
          input_data,
          target_data,
          weights_data,
          sizeAverage,
          THCudaTensor_size(state, input, 0),
          THCudaTensor_size(state, input, 1),
          n_classes
          );
  }

  hipError_t errcode = hipGetLastError();
  if (errcode != hipSuccess) {
    THError(hipGetErrorString(errcode));
  }
  if (weights) {
    THCudaTensor_free(state, weights);
  }
  THCudaTensor_free(state, target);
  THCudaTensor_free(state, input);
  return 0;
}

//void CudaClassNLLCriterionWithIgnore_updateGradInput(THCState *state, THCudaTensor *input, THCudaTensor *target, THCudaTensor *gradInput, bool sizeAverage, THCudaTensor *weights, THCudaTensor *total_weight) {
int CudaClassNLLCriterionWithIgnore_updateGradInput(lua_State* L) {
  THCState* state = getCutorchState(L);
  THCudaTensor* input = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
  THCudaTensor* target = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor* gradInput = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
  bool sizeAverage = (bool)luaL_checkint(L, 4);
  THCudaTensor* weights = (THCudaTensor*)luaT_checkudata(L, 5, "torch.CudaTensor");
  THCudaTensor* total_weight = (THCudaTensor*)luaT_checkudata(L, 6, "torch.CudaTensor");
  printf("%p\n", weights);
  printf("%p\n", total_weight);

  if (THCudaTensor_nDimension(state, target) > 1) {
    THError("multi-target not supported");
  }

  int n_dims = THCudaTensor_nDimension(state, input);
  int n_classes = THCudaTensor_size(state, input, n_dims - 1);

  THArgCheck(THCudaTensor_isContiguous(state, gradInput), 4, "gradInput must be contiguous");

  if (weights) {
    THAssert(THCudaTensor_checkGPU(
          state, 5, weights, input, target, gradInput, total_weight
          ));
  }
  else {
    THAssert(THCudaTensor_checkGPU(
          state, 4, input, target, gradInput, total_weight
          ));
  }

  if (THCudaTensor_nDimension(state, input) > 2) {
    THArgCheck(0, 2, "vector or matrix expected");
  }

  weights = weights ? THCudaTensor_newContiguous(state, weights) : NULL;
  target = THCudaTensor_newContiguous(state, target);

  float *weights_data = weights ? THCudaTensor_data(state, weights) : NULL;
  float *gradInput_data = THCudaTensor_data(state, gradInput);
  float *target_data = THCudaTensor_data(state, target);
  float *total_weight_data = THCudaTensor_data(state, total_weight);

  if (THCudaTensor_nDimension(state, input) == 1) {
    cunn_ClassNLLCriterionWithIgnore_updateGradInput_kernel1
      <<<1, 1, 0, THCState_getCurrentStream(state)>>>(
          gradInput_data,
          weights_data,
          target_data,
          total_weight_data,
          sizeAverage,
          n_classes
          );
  } else {
    cunn_ClassNLLCriterionWithIgnore_updateGradInput_kernel
      <<<1, NTHREADS, 0, THCState_getCurrentStream(state)>>>(
          gradInput_data,
          target_data,
          weights_data,
          total_weight_data,
          sizeAverage,
          THCudaTensor_size(state, input, 0),
          THCudaTensor_size(state, input, 1),
          n_classes
          );
  }
  hipError_t errcode = hipGetLastError();
  if (errcode != hipSuccess) {
    THError(hipGetErrorString(errcode));
  }
  if (weights) {
    THCudaTensor_free(state, weights);
  }
  THCudaTensor_free(state, target);
  return 0;
}


static const struct luaL_Reg funcs[] = {
  {"updateGradInput", CudaClassNLLCriterionWithIgnore_updateGradInput},
  //{"updateOutput", CudaClassNLLCriterionWithIgnore_updateOutput},
  {NULL, NULL}
};

extern "C" int luaopen_lib_ClassNLLCriterionWithIgnore(lua_State *L) {
  luaL_openlib(L, "lib_ClassNLLCriterionWithIgnore", funcs, 0);
  return 1;
}
